#include "hip/hip_runtime.h"
#include <iostream>

#include <Eigen/Core>
#include <Eigen/Eigenvalues>

#include "test_common.h"

using namespace Eigen;

__host__ __device__ void eigenValues(Matrix3d *m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret) {
#if TEST_DEBUG
  printf("Matrix(0,0): %f\n", (*m)(0, 0));
  printf("Matrix(1,1): %f\n", (*m)(1, 1));
  printf("Matrix(2,2): %f\n", (*m)(2, 2));
#endif
  SelfAdjointEigenSolver<Matrix3d> es;
  es.computeDirect(*m);
  (*ret) = es.eigenvalues();
  return;
}

__global__ void kernel(Matrix3d *m, Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret) {
  eigenValues(m, ret);
}

void testEigenvalues() {
  std::cout << "TEST EIGENVALUES" << std::endl;
  Matrix3d m;
  fillMatrix(m);
  m += m.transpose().eval();

  Matrix3d *m_gpu = nullptr;
  Matrix3d *mgpudebug = new Matrix3d();
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret =
      new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret1 =
      new Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType;
  Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType *ret_gpu = nullptr;
  eigenValues(&m, ret);
#if TEST_DEBUG
  std::cout << "Generated Matrix M 3x3:\n" << m << std::endl;
  std::cout << "The eigenvalues of M are:" << std::endl << (*ret) << std::endl;
  std::cout << "*************************\n\n" << std::endl;
#endif
  hipMalloc((void **)&m_gpu, sizeof(Matrix3d));
  hipMalloc((void **)&ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType));
  hipMemcpy(m_gpu, &m, sizeof(Matrix3d), hipMemcpyHostToDevice);

  kernel<<<1, 1>>>(m_gpu, ret_gpu);
  hipDeviceSynchronize();

  hipMemcpy(mgpudebug, m_gpu, sizeof(Matrix3d), hipMemcpyDeviceToHost);
  hipMemcpy(
      ret1, ret_gpu, sizeof(Eigen::SelfAdjointEigenSolver<Matrix3d>::RealVectorType), hipMemcpyDeviceToHost);
#if TEST_DEBUG
  std::cout << "GPU Generated Matrix M 3x3:\n" << (*mgpudebug) << std::endl;
  std::cout << "GPU The eigenvalues of M are:" << std::endl << (*ret1) << std::endl;
  std::cout << "*************************\n\n" << std::endl;
#endif
  assert(isEqualFuzzy(*ret, *ret1));
}

int main(int argc, char *argv[]) {

  testEigenvalues();

  return 0;
}